#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>

#define HEADER_SIZE 138
#define BLOCK_SIZE 16

typedef unsigned char BYTE;

__host__ __device__ float3 add_float3(float3 A, float3 B) {
    float3 C = {A.x + B.x, A.y + B.y, A.z + B.z};
    return C;
}

__host__ __device__ float3 sub_float3(float3 A, float3 B) {
    float3 C = {A.x - B.x, A.y - B.y, A.z - B.z};
    return C;
}

__host__ __device__ float dot_product(float3 v1, float3 v2) {
    return v1.x*v2.x + v1.y*v2.y + v1.z*v2.z;
}


void saveImage(int width, int height, float3** image, bool gpu) {
    char path[255] = "images/raytracing_cpu.txt";
    FILE *file = NULL; 
    file = fopen(path, "w");

    // fwrite(g_info, sizeof(BYTE), HEADER_SIZE, file); TODO uncomment me

    for (int h = 0; h < height; h++) {
        for (int w = 0; w < width; w++) {
            char pixel_x = (char)((image[h][w].x > 255.0f) ? 255.0f :
                                (image[h][w].x < 0.0f)   ? 0.0f :
                                image[h][w].x);
            char pixel_y = (char)((image[h][w].y > 255.0f) ? 255.0f :
                                (image[h][w].y < 0.0f)   ? 0.0f :
                                image[h][w].y);
            char pixel_z = (char)((image[h][w].z > 255.0f) ? 255.0f :
                                (image[h][w].z < 0.0f)   ? 0.0f :
                                image[h][w].z);

            fputc(pixel_x, file); // TODO change me
            fputc(pixel_y, file); // TODO change me
            fputc(pixel_z, file); // TODO change me
        }
    }

    fclose(file);
}

__host__ __device__ float3 normalize(float3 v) {
    float norma = sqrt(v.x*v.x + v.y*v.y + v.z*v.z);
    v.x /= norma;
    v.y /= norma;
    v.z /= norma;
    return v;
}

/* Return the distance from O to the intersection of the ray (O, D) with the sphere (S, R) 
    O and S are 3D points, D (direction) is a normalised vector and R is a scalar */
__host__ __device__ float intersect_sphere(float3 O, float3 D, float3 S, float R) {
    float a = dot_product(D, D);
    float3 OS = sub_float3(O, S);
    float b = 2 * dot_product(D, OS);
    float c = dot_product(OS, OS) - R * R;
    float disc = b*b - 4*a*c;
    if (disc > 0) {
        float discSqrt = sqrt(disc);
        float q = b < 0 ? (-b - discSqrt) / 2.0 : (-b + discSqrt) / 2.0;
        float t0 = q / a;
        float t1 = c / q;
        float tmin = min(t0, t1);
        float tmax = max(t0, t1);
        if (t1 >= 0) return t0 < 0 ? t1 : t0;
    }
    return INFINITY;
}

/* Find first point of intersection with the scene, trace a ray and apply Bling-phon shading */
__host__ __device__ float3 trace_ray(float3 O, float3 D, float3 position, float radius, float3 L, float ambient, 
            float diffuse, float3 color, float specular_c, int specular_k, float3 color_light) {
    float t = intersect_sphere(O, D, position, radius);
    if (t == INFINITY) return (float3){INFINITY, INFINITY, INFINITY}; // means no intersection


    float3 M = {O.x + D.x*t, O.y + D.y*t, O.z + D.z*t};

    float3 N = normalize(sub_float3(M, position));
    float3 toL = normalize(sub_float3(L, M));
    float3 toO = normalize(sub_float3(O, M));

    float3 col;
    float d_prod = max(dot_product(N, toL), 0.);
    col.x = ambient + diffuse * d_prod * color.x;
    col.y = ambient + diffuse * d_prod * color.y;
    col.z = ambient + diffuse * d_prod * color.z;

    d_prod = max(dot_product(N, normalize(add_float3(toL, toO))), 0.);
    col.x += specular_c * color_light.x * pow(d_prod, specular_k);
    col.y += specular_c * color_light.y * pow(d_prod, specular_k);
    col.z += specular_c * color_light.z * pow(d_prod, specular_k);

    return col;
}

float3** cpu_compute(int width, int height, float3 O, float3 Q, float3 position, float radius, float3 L, 
            float ambient, float diffuse, float3 color, float specular_c, int specular_k, float3 color_light) {
    float3 **img = (float3**)malloc(height * sizeof(float3*));
    for (int h = 0; h < height; h++) img[h] = (float3*)calloc(width, sizeof(float3));

    double step = 2. / (width - 1);
    int counterWidth = 0;
    int counterHeight = 0;
    for (double w = -1.; w < 1.005; w += step) {
        counterHeight = -1;
        for (double h = -1.; h < 1.005; h += step) {
            counterHeight++;
            Q.x = w, Q.y = h;
            float3 D = normalize(sub_float3(Q, O));
            float3 col = trace_ray(O, D, position, radius, L, ambient, diffuse, color, specular_c, specular_k, color_light);
            if (col.x == INFINITY) continue;

            col.x = col.x > 1 ? 1 : col.x < 0 ? 0 : col.x; 
            col.y = col.y > 1 ? 1 : col.y < 0 ? 0 : col.y; 
            col.z = col.z > 1 ? 1 : col.z < 0 ? 0 : col.z; 

            img[height - counterHeight - 1][counterWidth] = col;
        }
        counterWidth++;
    }
    return img;
}

__global__ void gpu_compute(int width, int height, float3 O, float3 Q, float3 position, float radius, float3 L, 
            float ambient, float diffuse, float3 color, float specular_c, int specular_k, float3 color_light){
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (!(x >= 0 && x < width && y >= 0 && y < height)) return;

    printf("X: %d\t Y: %d\n", x, y);
}

int main() {
    int width = 5, height = 5;

    // sphere properties
    float3 position = {0., 0., 1.};
    float radius = 1.;
    float3 color = {0., 0., 1.};
    float diffuse = 1.;
    float specular_c = 1.;
    int specular_k = 50;

    // light position and color
    float3 L = {5., 5., -10.};
    float3 color_light = {1., 1., 1.};
    float ambient = 0.05;

    // camera
    float3 O = {0., 0., -1.};
    float3 Q = {0., 0., 0.};

    // compute in cpu
    printf("Computing on CPU...\n\n");
    float3** img_cpu = cpu_compute(width, height, O, Q, position, radius, L, ambient, diffuse, color, specular_c, specular_k, color_light);
    
    // compute in gpu
    dim3 grid(((width  + (BLOCK_SIZE - 1)) / BLOCK_SIZE),
                      ((height + (BLOCK_SIZE - 1)) / BLOCK_SIZE));                       
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    printf("Computing on GPU...\n\n");
    gpu_compute<<<grid, block>>>(width, height, O, Q, position, radius, L, ambient, diffuse, color, specular_c, specular_k, color_light);
    hipDeviceSynchronize();
    // float3 **img_gpu = (float3**)malloc(height * sizeof(float3*));
    // for (int h = 0; h < height; h++) img_gpu[h] = (float3*)calloc(width, sizeof(float3));

    // hipHostMalloc(&img_gpu, sizeof())

    
    // float3** img_gpu = gpu_compute(width, height, O, Q, position, radius, L, ambient, diffuse, color, specular_c, specular_k, color_light);

    printf("Printing image...\n");

    saveImage(width, height, img_cpu, false);
    printf("Done!\n");
    return 0;
}